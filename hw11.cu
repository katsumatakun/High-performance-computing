
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
using namespace std;



__global__ void initArray( int *A) {
  int tid;
  tid = blockIdx.x * blockDim.x + threadIdx.x;
  A[tid] = tid;
}

__global__ void swapArray( int *A, int size, int num_t) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i=size/2/num_t*tid; i<size/2/num_t*(tid+1); i++){
    int temp = A[i];
    A[i] = A[size-1-i];
    A[size-1-i] = temp;
  }
}

__global__ void mmmul(int N, float *A, float *B, float *C){
  //printf("working");
  //printf("A[1]: %d", A[1]);
  int T = 8;
  __shared__ float smem_c[64][64];
  if(threadIdx.x==0 && threadIdx.y==0){
    for (int i=0; i<64; i++){
      for(int j=0; j<64; j++){
        smem_c[i][j]=0.0;
    }
  }
}

  __shared__ float smem_a[64][8];
  __shared__ float smem_b[8][64];
  //printf("%f", smem_c[0][0]);
  int c = blockIdx.x * 64;
  int r = blockIdx.y * 64;
  //printf("%d \n",blockDim.x);
  //printf("%d \n", blockDim.y);
  if(threadIdx.y==31){
	//printf("x: %d \n",threadIdx.x);
}
  int cthread = c + threadIdx.x * 2;
  int rthread = r + threadIdx.y * 2;
  if(threadIdx.x==1 && threadIdx.y==1){
     //printf("==========r, c; %d, %d\n", rthread, cthread);
}
  int count= 0;
  for (int kk=0; kk<N; kk+=T) {
    for (int i=threadIdx.x+blockDim.x*threadIdx.y; i<64*8; i+=blockDim.x*blockDim.y) {
    int k = kk +i / 64;
    int rt = r +i % 64;
    int ct = c +i % 64;
    smem_a[i%64][i/64] = A[rt*N+k];
    smem_b[i/64][i%64] = B[k*N+ct];
    //smem_c[i%64][i%64] = 0;
    //printf("%d %d \n", smem_a[i%64][i/64], smem_b[i/64][i%64]);
    //printf("%d %d \n", A[rt*N+k], B[k*N+ct]);
  }
  // if(threadIdx.x==0 && blockIdx.x==0 && threadIdx.y==0 && blockIdx.y==0){
  //   printf("K: %d",k);
  //   for (int i=0; i<64; i++){
  //     for(int j=0; j<8; j++){
  //       if(smem_a[i][j]!=1 || smem_b[j][i] !=1){
  //         printf("incorect, a: %d, b: %d\n",smem_a[i][j], smem_b[i][j]);
  //       }
  //     }
  //   }
  // }
    __syncthreads();
    //printf("finish loading %f \n", smem_c[0][0]);
    for (int x=0; x<2; x++){
      //printf("x loop \n");
      for (int y=0; y<2; y++){
        //printf("y loop");
        int cc = cthread % 64 + x;
        int rc = rthread % 64 + y;
if(threadIdx.x==0 && blockIdx.x==0 && threadIdx.y==0 && blockIdx.y==0)
        printf("rc,cc = %d,%d\n",rc,cc );
        for (int k=kk; k<kk+T; k++){

          int k8 = k % T;
          //int rc64 = rc % 64;
          //int cc64 = cc % 64;
        float temp = smem_a[rc][k8] * smem_b[k8][cc];

          atomicAdd(&smem_c[rc][cc],temp);
          //smem_c[rc][cc]+=temp;
          if(threadIdx.x==0 && blockIdx.x==0 && threadIdx.y==0 && blockIdx.y==0){
            //printf("temp=%f\n",temp );
            //printf("rc,cc=%f\n", smem_c[rc][cc]);
          }
          count+=1;
          //printf("count %d \n", count);
          //printf("load into c \n");

	}
      }
    }

    __syncthreads();
    if(threadIdx.x==0 && blockIdx.x==0 && threadIdx.y==0 && blockIdx.y==0){
      printf("K: %d\n",kk);
      for (int i=0; i<2; i++){
        for(int j=0; j<2; j++){
          printf("%f ", smem_c[i][j]);
        }
        printf("\n");
      }
    }
     __syncthreads();
  }
    __syncthreads();
    //printf("working \n");
    for (int p=0; p<2; p++){
    for (int q=0; q<2; q++) {
      int ccc = cthread + p;
      int rcc = rthread + q;
      /*if(smem_c[rcc%64][ccc%64]!=1024.0){
//if(rcc%64 != 31 && rcc%64 !=30 && rcc%64 !=29)
printf("row, col: %d %d \n", rcc, ccc);
}*/
      C[rcc*N+ccc] = smem_c[rcc%64][ccc%64];
    }
}
    __syncthreads();
    //printf("C %d\n", C[0]);


}

//double func(int N){
//   // int N = 1024;
//   srand(time(0));
//   int size=N*N;
//   //int size=16;
//   float* A = new float[size];
//   float* B = new float[size];
//   float* C = new float[size];
//   float* Cseq = new float[size];
//
//   int num;
//   for(int i=0; i<size; i++){
//     num = rand()%100;
//     A[i] = num;
//     B[i] = num;
//   }
//
// /*  for(int i=0; i<N; i++){
//     for(int j=0; j<N; j++){
//       for(int k=0; k<N; k++){
//         Cseq[i*N+j] += A[i*N+k]*B[k*N+j];
//       }
//     }
//   }*/
//
//
//
//   float *d_a;
//   float *d_b;
//   float *d_c;
//
//   // define thread hierarchy
//   int num_blocks_x = N/64; int num_blocks_y = N/64; int num_th_per_blk = 32;
//   // int num_t = num_blocks_x*num_blocks_y*num_th_per_blk;
//
//   // allocate host and device memory
//   size_t memSize;
//   // memSize = num_blocks * num_th_per_blk * sizeof(int);
//   memSize = size*sizeof(int);
//   // h_a = (int*) malloc(memSize);
//   clock_t start = clock();
//
//   cudaMalloc( (void**) &d_a, memSize);
//   cudaMemcpy( d_a, A, memSize, cudaMemcpyHostToDevice);
//   cudaMalloc( (void**) &d_b, memSize);
//   cudaMemcpy( d_b, B, memSize, cudaMemcpyHostToDevice);
//   cudaMalloc( (void**) &d_c, memSize);
//   cudaMemcpy( d_c, C, memSize, cudaMemcpyHostToDevice);
//
//   // launch kernel
//   dim3 dimGrid2D(num_blocks_x, num_blocks_y);
//   dim3 dimBlock(num_th_per_blk, num_th_per_blk);
//   mmmul<<< dimGrid2D, dimBlock >>>(N, d_a, d_b, d_c);
//   cudaMemcpy( C, d_c, memSize, cudaMemcpyDeviceToHost);
//   clock_t end = clock();
//
//   double time_elapsed_in_seconds = (double)(end - start)/CLOCKS_PER_SEC;
//
//
//
//   /*for(int i=0; i<N*N; i+=1){
//     if(C[i]!=Cseq[i]){
//     printf("%f %f %d\n",Cseq[i],C[i], i);
// }
// }*/

// return time_elapsed_in_seconds;

//}


int main(int argc, char *argv[]){

  int N = 1024*4;
  srand(time(0));
  int size=N*N;
  //int size=16;
  float* A = new float[size];
  float* B = new float[size];
  float* C = new float[size];
  float* Cseq = new float[size];

  float num;
  for(int i=0; i<size; i++){
    num = rand()%10;
    A[i] = num;
    B[i] = num;
  }

for(int i=0; i<N; i++){
    for(int j=0; j<N; j++){
      for(int k=0; k<N; k++){
        Cseq[i*N+j] += A[i*N+k]*B[k*N+j];
      }
    }
  }



  float *d_a;
  float *d_b;
  float *d_c;

  // define thread hierarchy
  int num_blocks_x = N/64; int num_blocks_y = N/64; int num_th_per_blk = 32;
  // int num_t = num_blocks_x*num_blocks_y*num_th_per_blk;

  // allocate host and device memory
  size_t memSize;
  // memSize = num_blocks * num_th_per_blk * sizeof(int);
  memSize = size*sizeof(int);
  // h_a = (int*) malloc(memSize);
  clock_t start = clock();

  hipMalloc( (void**) &d_a, memSize);
  hipMemcpy( d_a, A, memSize, hipMemcpyHostToDevice);
  hipMalloc( (void**) &d_b, memSize);
  hipMemcpy( d_b, B, memSize, hipMemcpyHostToDevice);
  hipMalloc( (void**) &d_c, memSize);
  hipMemcpy( d_c, C, memSize, hipMemcpyHostToDevice);

  // launch kernel
  dim3 dimGrid2D(num_blocks_x, num_blocks_y);
  dim3 dimBlock(num_th_per_blk, num_th_per_blk);
  mmmul<<< dimGrid2D, dimBlock >>>(N, d_a, d_b, d_c);
  hipMemcpy( C, d_c, memSize, hipMemcpyDeviceToHost);
  clock_t end = clock();

  double time_elapsed_in_seconds = (double)(end - start)/CLOCKS_PER_SEC;



  for(int i=0; i<N*N; i+=1){
    if(C[i]!=Cseq[i]){
    printf("%f %f %d\n",Cseq[i],C[i], i);
}
//if(C[i]!= 4096.0){
//printf("C[i]:%f, i:%d", C[i], i);
//}
}

  // for(int i=1; i<3; i++){
    printf("execution time for N = %d was %f\n", N, time_elapsed_in_seconds);
  // }
  return 0;
}
