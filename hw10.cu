
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
using namespace std;



__global__ void initArray( int *A) {
  int tid;
  tid = blockIdx.x * blockDim.x + threadIdx.x;
  A[tid] = tid;
}

__global__ void swapArray( int *A, int size) {

  for(int i=0; i<size/2; i++){
    int temp = A[i];
    A[i] = A[size-1-i];
    A[size-1-i] = temp;
  }
}


int main(){

  srand(time(0));
  int size=16*1024*1024;
  // int size=16;
  int* ary = new int[size];
  int* ans = new int[size];
  int num;
  for(int i=0; i<size; i++){
    num = rand()%100;
    ary[i] = num;
    ans[size-i-1] = num;
  }

  int *h_a, *d_a;
  // define thread hierarchy
  int num_blocks = 8; int num_th_per_blk = 8;
  // allocate host and device memory
  size_t memSize;
  // memSize = num_blocks * num_th_per_blk * sizeof(int);
  memSize = size*sizeof(int);
  // h_a = (int*) malloc(memSize);
  hipMalloc( (void**) &d_a, memSize);

  hipMemcpy( d_a, ary, memSize, hipMemcpyHostToDevice);
  // launch kernel
  dim3 dimGrid(num_blocks);
  dim3 dimBlock(num_th_per_blk);
  // initArray<<< dimGrid, dimBlock >>>(d_a);
  swapArray<<< dimGrid, dimBlock >>>(d_a, size);
  // retrieve results
  hipMemcpy( ary, d_a, memSize, hipMemcpyDeviceToHost);

  for(int i=0; i<size; i++){
    if(ary[i]!=ans[i]){
      cout << ans[i] << " "<< ary[i] << endl;
      cout << "not match" << endl;
      break;
  }
  
}

  return 0;
}
