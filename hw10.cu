
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
using namespace std;



__global__ void initArray( int *A) {
  int tid;
  tid = blockIdx.x * blockDim.x + threadIdx.x;
  A[tid] = tid;
}

__global__ void swapArray( int *A, int size, int num_t) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i=size/2/num_t*tid; i<size/2/num_t*(tid+1); i++){
    int temp = A[i];
    A[i] = A[size-1-i];
    A[size-1-i] = temp;
  }
}


int main(){

  srand(time(0));
  int size=16*1024*1024;
  //int size=16;
  int* ary = new int[size];
  int* ans = new int[size];
  int num;
  for(int i=0; i<size; i++){
    num = rand();
    ary[i] = num;
    ans[size-i-1] = num;
  }

  int *d_a;
  // define thread hierarchy
  int num_blocks = 8; int num_th_per_blk = 16;
  int num_t = num_blocks*num_th_per_blk;

  // allocate host and device memory
  size_t memSize;
  // memSize = num_blocks * num_th_per_blk * sizeof(int);
  memSize = size*sizeof(int);
  // h_a = (int*) malloc(memSize);
  hipMalloc( (void**) &d_a, memSize);

  hipMemcpy( d_a, ary, memSize, hipMemcpyHostToDevice);
  // launch kernel
  dim3 dimGrid(num_blocks);
  dim3 dimBlock(num_th_per_blk);
 
  swapArray<<< dimGrid, dimBlock >>>(d_a, size, num_t);
  // retrieve results
  hipMemcpy( ary, d_a, memSize, hipMemcpyDeviceToHost);

  for(int i=0; i<size; i++){
    if(ary[i]!=ans[i]){
      cout << i << " "<< ans[i] << " "<< ary[i] << endl;
      cout << "not match" << endl;
      break;
  }
//	cout << ans[i] << " "<<ary[i] <<endl;

}

  return 0;
}
